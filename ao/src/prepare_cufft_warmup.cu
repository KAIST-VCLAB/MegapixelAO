#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>
#include <hip/hip_complex.h>
#include <hip/hip_runtime_api.h>
#include <assert.h>

typedef hipfftComplex complex;

void cufft_warper(complex *h_in, int n, int m, hipfftHandle plan, complex *h_out)
{
    const int data_size = n*m*sizeof(complex);

    // device memory allocation
    complex *d_temp;
    checkCudaErrors(hipMalloc(&d_temp,  data_size));

    // transfer data from host to device
    checkCudaErrors(hipMemcpy(d_temp, h_in, data_size, hipMemcpyHostToDevice));

	// Compute the FFT
	hipfftExecC2C(plan, d_temp, d_temp, HIPFFT_FORWARD);

    // transfer result from device to host
    checkCudaErrors(hipMemcpy(h_out, d_temp, data_size, hipMemcpyDeviceToHost));

    // cleanup
    checkCudaErrors(hipFree(d_temp));
}